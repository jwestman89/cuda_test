#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addArrays(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    const int n = 10;
    int a[n], b[n], c[n];

    // Initialize input arrays
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, n * sizeof(int));
    hipMalloc((void **) &d_b, n * sizeof(int));
    hipMalloc((void **) &d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    addArrays<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < n; i++)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}